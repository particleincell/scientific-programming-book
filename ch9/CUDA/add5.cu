#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <chrono>
using namespace std;

using type = float;
#define CUDA_ERROR(f) {if (f!=hipSuccess) {cerr<<hipGetErrorString(f)<<" on line "<<__LINE__<<endl;exit(-1);}}
__global__ void add(type *a, type *b, type *c, const int N)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if (i<N)
		c[i] = a[i]+b[i];	
}
 
int main()
{
	hipDeviceProp_t cuda_props;

	CUDA_ERROR(hipGetDeviceProperties(&cuda_props,0));
	if (!cuda_props.deviceOverlap) {
		cerr<<"Your GPU does not support concurrent copy and execute kernels"<<endl;
		return -1;
		}

	/*allocate vectors*/
	const int N = 50000000;		/*50 million*/
	
	type *a;
	type *b;
	type *c;

	/*allocated pinned memory on the CPU*/
	CUDA_ERROR(hipHostAlloc(&a,sizeof(type)*N,hipHostMallocDefault));
	CUDA_ERROR(hipHostAlloc(&b,sizeof(type)*N,hipHostMallocDefault));
	CUDA_ERROR(hipHostAlloc(&c,sizeof(type)*N,hipHostMallocDefault));

	type *dev_a;		/*pointer to data on the GPU*/
	type *dev_b;		/*pointer to data on the GPU*/
	type *dev_c;		/*pointer to data on the GPU*/
	
	/*allocate memory on the GPU*/
	CUDA_ERROR(hipMalloc((void**)&dev_a, N*sizeof(type)));
	CUDA_ERROR(hipMalloc((void**)&dev_b, N*sizeof(type)));
	CUDA_ERROR(hipMalloc((void**)&dev_c, N*sizeof(type)));

	/*compute number of blocks using chunk size*/
	int threads_per_block = 512;
	int num_blocks = (N+threads_per_block-1)/threads_per_block;

	/*starting time for no-stream case*/
	auto t1 = chrono::high_resolution_clock::now();	

	/*copy a and b vectors*/
	CUDA_ERROR(hipMemcpy(dev_a,a,N*sizeof(type),hipMemcpyHostToDevice));
	CUDA_ERROR(hipMemcpy(dev_b,b,N*sizeof(type),hipMemcpyHostToDevice));
		
	/*launch kernel execution*/
	add<<<num_blocks, threads_per_block>>>(dev_a, dev_b, dev_c, N);
	CUDA_ERROR(hipPeekAtLastError());

	/*copy back*/
	CUDA_ERROR(hipMemcpy(c,dev_c,N*sizeof(type),hipMemcpyDeviceToHost));
	auto t2 = chrono::high_resolution_clock::now();	

    std::chrono::duration<double, std::milli> duration1 = t2-t1;
	cout<<"GPU time with pinned memory "<<duration1.count()<<" ms"<<endl;
	
	bool agree = true;
	for (int i=0;i<N;i++) if (a[i]+b[i]!=c[i]) {agree=false;break;}
	cout<<"Results "<<(agree?"":"DO NOT ")<<"agree"<<endl;

	/*Part 2: streams*/
		
	/*create streams*/
	hipStream_t stream1,stream2;
	CUDA_ERROR(hipStreamCreate(&stream1));
	CUDA_ERROR(hipStreamCreate(&stream2));
	
	const int CHUNK = N/10;
	
	/*recompute number of blocks using chunk size*/
	num_blocks = (CHUNK+threads_per_block-1)/threads_per_block;

	/*starting time*/
	auto t3 = chrono::high_resolution_clock::now();	

	for (int i=0;i<N;i+=CHUNK*2)
	{
		/*schedule copies of chunk of "a" on streams 1 and 2*/
		CUDA_ERROR(hipMemcpyAsync(dev_a,a+i,CHUNK*sizeof(type),hipMemcpyHostToDevice,stream1));
		CUDA_ERROR(hipMemcpyAsync(dev_a,a+i+CHUNK,CHUNK*sizeof(type),hipMemcpyHostToDevice,stream2));

		/*repeat for vector "b"*/
		CUDA_ERROR(hipMemcpyAsync(dev_b,b+i,CHUNK*sizeof(type),hipMemcpyHostToDevice,stream1));
		CUDA_ERROR(hipMemcpyAsync(dev_b,b+i+CHUNK,CHUNK*sizeof(type),hipMemcpyHostToDevice,stream2));

		/*schedule kernel execution*/
		add<<<num_blocks, threads_per_block,0,stream1>>>(dev_a, dev_b, dev_c, CHUNK);
		add<<<num_blocks, threads_per_block,0,stream2>>>(dev_a, dev_b, dev_c, CHUNK);
		CUDA_ERROR(hipPeekAtLastError());

		/*schedule memory copy back*/
		CUDA_ERROR(hipMemcpyAsync(c+i,dev_c,CHUNK*sizeof(type),hipMemcpyDeviceToHost,stream1));
		CUDA_ERROR(hipMemcpyAsync(c+i+CHUNK,dev_c,CHUNK*sizeof(type),hipMemcpyDeviceToHost,stream2));
	}

	/*wait for code to finish running*/
	CUDA_ERROR(hipStreamSynchronize(stream1));
	CUDA_ERROR(hipStreamSynchronize(stream2));

	auto t4 = chrono::high_resolution_clock::now();	

    std::chrono::duration<double, std::milli> duration2 = t4-t3;
	cout<<"GPU time with pinned memory and streams "<<duration2.count()<<" ms"<<endl;

	/*make sure results are correct*/
	agree = true;
	for (int i=0;i<N;i++) if (a[i]+b[i]!=c[i]) {agree=false;break;}
	cout<<"Results "<<(agree?"":"DO NOT ")<<"agree"<<endl;

	/*free CPU memory*/
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);

	/*free GPU memory*/
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);

	CUDA_ERROR(hipDeviceReset());

	return 0;
}

